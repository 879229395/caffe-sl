#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void L2NormLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const int count = top[0]->count();
  Dtype* top_data = top[0]->mutable_gpu_data();
  caffe_gpu_dot(count, bottom[0]->gpu_data(), bottom[0]->gpu_data(), &this->lpnorm_);
  this->lpnorm_ = std::sqrt(this->lpnorm_);
  caffe_gpu_scale(count, this->lpnorm_, bottom[0]->gpu_data(), top_data);
}

template <typename Dtype>
void L2NormLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const int count = top[0]->count();
  const Dtype* top_diff = top[0]->gpu_diff();
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

    CHECK_EQ(0, 1) << "WRONG Impl !!!";
    caffe_gpu_scale(count, (Dtype)(1.0 / this->lpnorm_), top_diff, bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(L2NormLayer);


}  // namespace caffe
